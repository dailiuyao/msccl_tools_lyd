#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

struct LogMessage_lyd* d_messages;

#if PROFILE_LYD_SEND_RECV_CHUNK == 1
std::chrono::time_point<std::chrono::high_resolution_clock> netIsend_time_start;
std::chrono::time_point<std::chrono::high_resolution_clock> netIrecv_time_start;
std::chrono::time_point<std::chrono::high_resolution_clock> netIsend_time_end;
std::chrono::time_point<std::chrono::high_resolution_clock> netIrecv_time_end;  
#endif

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}

uint64_t rdtsc() {
    uint32_t lo, hi;
    // Inline assembly to read the TSC
    __asm__ __volatile__ ("rdtsc" : "=a" (lo), "=d" (hi));
    return (uint64_t)hi << 32 | lo;
}

int main(int argc, char* argv[])
{

  const char* env_gauge_heo_var = getenv("GAUGE_HEO");

  const char* env_gauge_mode_var = getenv("GAUGE_MODE");

  const char* env_gauge_iteration_var = getenv("GAUGE_ITERATION");

  const char* env_gauge_nchannels_var = getenv("GAUGE_NCHANNELS");

  const char* env_gauge_chunk_size_var = getenv("GAUGE_CHUNK_SIZE");

  const char* env_gauge_output_dir_var = getenv("GAUGE_OUT_DIRE");

  const char* env_gauge_nthreads_var = getenv("NCCL_NTHREADS");

  const char* env_comm_gpu_id_var = getenv("COMM_GPU_ID");

  // Check if environment variables are set
  if (!env_gauge_heo_var) env_gauge_heo_var = "unknown_gauge_heo";
  if (!env_gauge_mode_var) env_gauge_mode_var = "unknown_gauge_mode";
  if (!env_gauge_iteration_var) env_gauge_iteration_var = "unknown_gauge_iteration";
  if (!env_gauge_nchannels_var) env_gauge_nchannels_var = "unknown_gauge_nchannels";
  if (!env_gauge_chunk_size_var) env_gauge_chunk_size_var = "unknown_gauge_chunk_size";
  if (!env_gauge_nthreads_var) env_gauge_nthreads_var = "unknown_gauge_nthreads";  
  if (!env_gauge_output_dir_var) {
    env_gauge_output_dir_var = "unknown_gauge_output_dir";
    printf("unknown gauge output dir\n");
  }

  long long size = 1;  // Default size
  const char* env_gauge_size_var = getenv("GAUGE_MESSAGE_SIZE");
  if (env_gauge_size_var != nullptr) {
      size = atoll(env_gauge_size_var) * 1024 / 4;  // Convert from kilobytes to number of floats, assuming the environment variable is in kilobytes
  }

  const char* env_gauge_step_var = getenv("GAUGE_STEP_SIZE");

  int gauge_step = atoi(env_gauge_step_var);

  int comm_gpu_id = atoi(env_comm_gpu_id_var);

  int N_CHUNKS;

  // if (gauge_step != 0) {
  //   if (gauge_step >= 16384) {
  //     N_CHUNKS = 128;
  //   } else {
  //     N_CHUNKS = atoi(env_gauge_size_var)/atoi(env_gauge_step_var); 
  //   }
  // } else {
  //   N_CHUNKS = 1;
  // }

  // if (N_CHUNKS == 0) N_CHUNKS = 1;

  N_CHUNKS = MAXLOGLYD;

  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  char filename[256];

  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }

  // select gpu on each node
  if (comm_gpu_id == 0) {
    localRank = localRank + 1;
  } else if (comm_gpu_id == 1){
    if (myRank == 0) localRank = localRank + 1;
  } else if (comm_gpu_id == 2){
    if (myRank == 1) localRank = localRank + 1;
  } else if (comm_gpu_id == 3){
    localRank = localRank + 3;
  } 


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, N_ITERS * size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, N_ITERS * size * sizeof(float))); 
  CUDACHECK(hipStreamCreate(&s));
  

  //gauge test
  CUDACHECK(hipMalloc(&d_messages, sizeof(LogMessage_lyd)));
  CUDACHECK(hipMemset(d_messages, 0, sizeof(LogMessage_lyd)));

  ////////////////////////////// PROFILE_LYD_P2P_DEVICE_SYNC: START //////////////////////////////
  
  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

  //communicating using NCCL

  hipEvent_t start, stop;
  float elapsed_time;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  #if PROFILE_LYD_SEND_RECV_CHUNK == 1 
  netIsend_time_start = std::chrono::time_point<std::chrono::high_resolution_clock>();
  netIrecv_time_start = std::chrono::time_point<std::chrono::high_resolution_clock>();
  netIsend_time_end = std::chrono::time_point<std::chrono::high_resolution_clock>();
  netIrecv_time_end = std::chrono::time_point<std::chrono::high_resolution_clock>();
  #endif

  CUDACHECK(hipStreamSynchronize(s));

  hipEventRecord(start, s);

  std::chrono::time_point<std::chrono::high_resolution_clock> nccl_func_start_time = std::chrono::high_resolution_clock::now(); 

  CUDACHECK(hipStreamSynchronize(s));

  //communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum,
        comm, s));

  CUDACHECK(hipStreamSynchronize(s));

  hipEventRecord(stop, s);

  std::chrono::time_point<std::chrono::high_resolution_clock> nccl_func_end_time = std::chrono::high_resolution_clock::now();

  // Wait for the stop event to complete
  hipEventSynchronize(stop);

  // Calculate elapsed time between events
  hipEventElapsedTime(&elapsed_time, start, stop);

  // Destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop); 

  #if PROFILE_LYD_SEND_RECV_CHUNK == 1 
  std::chrono::duration<float, std::milli> func_netIsend_time = netIsend_time_start - nccl_func_start_time; 

  std::chrono::duration<float, std::milli> netIsend_total_time = netIsend_time_end - netIsend_time_start;  

  std::chrono::duration<float, std::milli> netIrecv_total_time = netIrecv_time_end - netIrecv_time_start;  

  std::chrono::duration<float, std::milli> netIrecv_func_time = nccl_func_end_time - netIrecv_time_end; 
  #endif
  
  std::chrono::duration<float, std::milli> nccl_func_time = nccl_func_end_time - nccl_func_start_time; 

  ////////////////////////////// PROFILE_LYD_P2P_DEVICE_SYNC: END //////////////////////////////

  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  if (myRank < 4) {
    sprintf(filename, "%s/nccl_allreduce_%s_r-%d.out", env_gauge_output_dir_var, env_gauge_heo_var, myRank);
    freopen(filename, "a", stdout);
  } else {
    freopen("/dev/null", "w", stdout);
  }

  // After the kernel execution, copy the messages back to the host
  LogMessage_lyd* h_messages = new LogMessage_lyd;
  hipMemcpy(h_messages, d_messages, sizeof(LogMessage_lyd), hipMemcpyDeviceToHost);

  #if PROFILE_LYD_REDUCE_BROADCAST == 1
  double gauge_time;
  
  if (myRank == 0) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    gauge_time = static_cast<double>(h_messages->timeValue[0][1] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("rrc_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, env_gauge_iteration_var, gauge_time);
  } else if (myRank == 1) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    gauge_time = static_cast<double>(h_messages->timeValue[0][1] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("send_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, env_gauge_iteration_var, gauge_time);
  } else if (myRank == 2) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    gauge_time = static_cast<double>(h_messages->timeValue[0][1] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("rrs_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, env_gauge_iteration_var, gauge_time);
  } else if (myRank == 3) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    gauge_time = static_cast<double>(h_messages->timeValue[0][1] - h_messages->timeValue[0][0]) / GAUGE_GPU_FREQUENCY;
    printf("send_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, env_gauge_iteration_var, gauge_time);
  }
  #endif

  #if PROFILE_LYD_REDUCE_BROADCAST_CHUNK == 1
  double gauge_time;
  
  if (myRank == 0) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    for (size_t i = 0; i < N_CHUNKS; ++i) { 
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[0][i]) / GAUGE_GPU_FREQUENCY;
      printf("rrc_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[3][i] - h_messages->timeValue[2][i]) / GAUGE_GPU_FREQUENCY;
      printf("send_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    }
  } else if (myRank == 1) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    for (size_t i = 0; i < N_CHUNKS; ++i) { 
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[0][i]) / GAUGE_GPU_FREQUENCY;
      printf("send_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[3][i] - h_messages->timeValue[2][i]) / GAUGE_GPU_FREQUENCY;
      printf("recv_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    }
  } else if (myRank == 2) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    for (size_t i = 0; i < N_CHUNKS; ++i) { 
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[0][i]) / GAUGE_GPU_FREQUENCY;
      printf("rrs_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[3][i] - h_messages->timeValue[2][i]) / GAUGE_GPU_FREQUENCY;
      printf("rcs_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    }
  } else if (myRank == 3) {
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
    printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
    for (size_t i = 0; i < N_CHUNKS; ++i) { 
      gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[0][i]) / GAUGE_GPU_FREQUENCY;
      printf("send_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
      gauge_time = static_cast<double>(h_messages->timeValue[3][i] - h_messages->timeValue[2][i]) / GAUGE_GPU_FREQUENCY;
      printf("recv_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    }
  }
  #endif

  #if PROFILE_LYD_RS_AG_CHUNK == 1
  double gauge_time;
  
  printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time: %f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, elapsed_time);
  printf("message size(%s)_nchannels(%s)_nthreads(%s)_n(%d)_iteration(%s)_nccl allreduce elapsed time by clock: %.3f ms\n", env_gauge_size_var, env_gauge_nchannels_var, env_gauge_nthreads_var, N_ITERS, env_gauge_iteration_var, nccl_func_time.count());
  for (size_t i = 0; i < N_CHUNKS; ++i) { 
    gauge_time = static_cast<double>(h_messages->timeValue[1][i] - h_messages->timeValue[0][i]) / GAUGE_GPU_FREQUENCY;
    printf("send_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[3][i] - h_messages->timeValue[2][i]) / GAUGE_GPU_FREQUENCY;
    printf("rrs_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[5][i] - h_messages->timeValue[4][i]) / GAUGE_GPU_FREQUENCY;
    printf("rrcs_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[7][i] - h_messages->timeValue[6][i]) / GAUGE_GPU_FREQUENCY;
    printf("rcs_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
    gauge_time = static_cast<double>(h_messages->timeValue[9][i] - h_messages->timeValue[8][i]) / GAUGE_GPU_FREQUENCY;
    printf("recv_heo(%s)_mode(%s)_nchannels(%s)_nthreads(%s)_chunk steps(%s)_message size(%s)_n(%d)_iteration(%s): %f us\n", env_gauge_heo_var, env_gauge_mode_var, env_gauge_nchannels_var, env_gauge_nthreads_var, env_gauge_chunk_size_var, env_gauge_size_var, i, env_gauge_iteration_var, gauge_time);
  }
  
  #endif

  // Free the device memory of the gauge test
  hipFree(d_messages);
  delete[] h_messages;


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);

  //finalizing MPI
  MPICHECK(MPI_Finalize());

  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}