#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

struct LogMessage_lyd* d_messages;
// int nccl_gauge_iteration = 0;
#define N_ITERS 20
#define MESSAGE_SIZE_GAUGE 8192
#define MESSAGE_SIZE_GAUGE_REAL MESSAGE_SIZE_GAUGE*4

#define GAUGE_OUT_DIRE "/home/yuke/ncclPG/CCL-LYD/msccl_tools_lyd/examples/scripts/ncclguage"

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


int main(int argc, char* argv[])
{
  int size = MESSAGE_SIZE_GAUGE*1024;
  // int size = 1;


  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));

  char filename[256];

  if (myRank < 2) {
    sprintf(filename, "%s/nccl-%d.out", GAUGE_OUT_DIRE, myRank);
    freopen(filename, "w", stdout);
  } else {
    freopen("/dev/null", "w", stdout);
  }


  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));
  

  //gauge test
  CUDACHECK(hipMalloc(&d_messages, sizeof(LogMessage_lyd)));
  CUDACHECK(hipMemset(d_messages, 0, sizeof(LogMessage_lyd)));

  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));


  //communicating using NCCL
  
  //P2P
  int recvPeer = (myRank-1+nRanks) % nRanks;
  int sendPeer = (myRank+1) % nRanks;

  // // Declare CUDA events
  // hipEvent_t start_0, stop_0;
  // hipEventCreate(&start_0);
  // hipEventCreate(&stop_0);
  // float milliseconds_0 = 0;

  // hipEventRecord(start_0, s);

  for (int i = 0 ; i < N_ITERS; i++) {
    NCCLCHECK(ncclGroupStart());
    if (myRank == 0) {
      NCCLCHECK(ncclSend((const void*)sendbuff, size, ncclFloat, sendPeer, comm, s));
    } else {
      NCCLCHECK(ncclRecv((void*)recvbuff, size, ncclFloat, recvPeer, comm, s));
    }
    NCCLCHECK(ncclGroupEnd());
  }

  NCCLCHECK(ncclGroupStart());
  if (myRank == 1) {
    NCCLCHECK(ncclSend((const void*)sendbuff, size, ncclFloat, sendPeer, comm, s));
  } else {
    NCCLCHECK(ncclRecv((void*)recvbuff, size, ncclFloat, recvPeer, comm, s));
  }
  NCCLCHECK(ncclGroupEnd());

  // hipEventRecord(stop_0, s);

  // hipEventSynchronize(stop_0);

  // hipEventElapsedTime(&milliseconds_0, start_0, stop_0);

  // printf("Rank %d | time: %f milliseconds\n", myRank, milliseconds_0);

  // // Clean up
  // hipEventDestroy(start_0);
  // hipEventDestroy(stop_0);


  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  // After the kernel execution, copy the messages back to the host
  LogMessage_lyd* h_messages = new LogMessage_lyd;
  hipMemcpy(h_messages, d_messages, sizeof(LogMessage_lyd), hipMemcpyDeviceToHost);

  // Process and print the messages on the host
  #if PROFILE_LYD_REDUCE_BROADCAST == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | allreduce.h | runTreeUpDown | recvReduceCopy | time: %f us\n", h_messages->timeValue[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | allreduce.h | runTreeUpDown | directSendFromOutput | time: %f us\n", h_messages->timeValue1[i][0]);
  }
  #endif

  #if PROFILE_LYD_REDUCE_BROADCAST_CHUNK == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | allreduce.h | runTreeUpDown | recvReduceCopy-chunk | iteration %d | time: %f us\n", j, h_messages->timeValue[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | allreduce.h | runTreeUpDown | directSendFromOutput-chunk | iteration %d | time: %f us\n", j, h_messages->timeValue1[i][j]);
    }
  }
  #endif

  #if PROFILE_LYD_REDUCE_LOADCONN_SETDATA == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | loadRecvConn | time: %f us\n", h_messages->timeValue[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | loadSendConn | time: %f us\n", h_messages->timeValue1[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | setDataPtrs | time: %f us\n", h_messages->timeValue2[i][0]);
  }
  #endif

  #if PROFILE_LYD_GENERIC == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | genericop | time: %f us\n", h_messages->timeValue[i][0]);
  }
  #endif

  #if PROFILE_LYD_WAIT_REDUCE_COPY_POST == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | waitpeer | iteration %d | time: %f us\n", j, h_messages->timeValue[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | ReduceOrCopyMulti | iteration %d | time: %f us\n", j, h_messages->timeValue1[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | postPeer | iteration %d | time: %f us\n", j, h_messages->timeValue2[i][j]);
    }
  }
  #endif

  #if PROFILE_LYD_SEND_RECV_CHUNK == 1
  // if (myRank == 0){
  //   printf("DEVICE | sendrecv.h | full recv - send | time: %f us\n", h_messages->timeValue2[0][3]-h_messages->timeValue2[0][0]);
  //   for (size_t i = 0; i < maxMessages; ++i) {
  //     for (size_t j = 0; j < MAXLOGLYD; j++){
  //       if (j>0) printf("DEVICE | sendrecv.h | runsend%d - runsend0 | warp %d | iteration %d | time: %f us\n", j, i, j, h_messages->timeValue[i][j] - h_messages->timeValue[i][0]);
  //       printf("DEVICE | sendrecv.h | runrecv - runsend | warp %d | iteration %d | time: %f us\n", i, j, h_messages->timeValue1[i][j] - h_messages->timeValue[i][j]);
  //     }
  //   }
  // } else {
  //   printf("DEVICE | sendrecv.h | full send - recv | time: %f us\n", h_messages->timeValue2[0][0]-h_messages->timeValue2[0][3]);
  //   for (size_t i = 0; i < maxMessages; ++i) {
  //     for (size_t j = 0; j < MAXLOGLYD; j++){
  //       printf("DEVICE | sendrecv.h | runsend - runrecv | warp %d | iteration %d | time: %f us\n", i, j, h_messages->timeValue[i][j] - h_messages->timeValue1[i][j]);
  //     }
  //   }
  // }

  if (myRank == 0) { 
    printf("DEVICE | sendrecv.h | runrecv - runsend | PRTT(%d, 0, %dKB) | time: %f us\n", N_ITERS, MESSAGE_SIZE_GAUGE_REAL, h_messages->timeValue[1][0] - h_messages->timeValue[0][0]);
    for (size_t i = 0; i < N_ITERS-1; ++i) {
      printf("DEVICE | sendrecv.h | runsend_%d - runsend_0 | time: %f us\n", i, h_messages->timeValue[0][i] - h_messages->timeValue[0][0]);
      printf("DEVICE | sendrecv.h | runsend_stop_%d - runsend_0 | time: %f us\n", i, h_messages->timeValue[2][i] - h_messages->timeValue[0][0]);
    }
  }
  #endif

  // Free the device memory of the gauge test
  hipFree(d_messages);
  delete[] h_messages;


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}