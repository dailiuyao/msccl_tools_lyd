#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>

struct LogMessage_lyd* d_messages;

#define MPICHECK(cmd) do {                          \
  int e = cmd;                                      \
  if( e != MPI_SUCCESS ) {                          \
    printf("Failed: MPI error %s:%d '%d'\n",        \
        __FILE__,__LINE__, e);   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",             \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",             \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


static uint64_t getHostHash(const char* string) {
  // Based on DJB2a, result = result * 33 ^ char
  uint64_t result = 5381;
  for (int c = 0; string[c] != '\0'; c++){
    result = ((result << 5) + result) ^ string[c];
  }
  return result;
}


static void getHostName(char* hostname, int maxlen) {
  gethostname(hostname, maxlen);
  for (int i=0; i< maxlen; i++) {
    if (hostname[i] == '.') {
        hostname[i] = '\0';
        return;
    }
  }
}


int main(int argc, char* argv[])
{
  int size = 32*1024*1024;


  int myRank, nRanks, localRank = 0;


  //initializing MPI
  MPICHECK(MPI_Init(&argc, &argv));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));


  //calculating localRank based on hostname which is used in selecting a GPU
  uint64_t hostHashs[nRanks];
  char hostname[1024];
  getHostName(hostname, 1024);
  hostHashs[myRank] = getHostHash(hostname);
  MPICHECK(MPI_Allgather(MPI_IN_PLACE, 0, MPI_DATATYPE_NULL, hostHashs, sizeof(uint64_t), MPI_BYTE, MPI_COMM_WORLD));
  for (int p=0; p<nRanks; p++) {
     if (p == myRank) break;
     if (hostHashs[p] == hostHashs[myRank]) localRank++;
  }


  ncclUniqueId id;
  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;


  //get NCCL unique ID at rank 0 and broadcast it to all others
  if (myRank == 0) ncclGetUniqueId(&id);
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


  //picking a GPU based on localRank, allocate device buffers
  CUDACHECK(hipSetDevice(localRank));
  CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK(hipStreamCreate(&s));
  

  //gauge test
  CUDACHECK(hipMalloc(&d_messages, sizeof(LogMessage_lyd)));
  CUDACHECK(hipMemset(d_messages, 0, sizeof(LogMessage_lyd)));

  //initializing NCCL
  NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));


  //communicating using NCCL
  NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum,
        comm, s));


  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK(hipStreamSynchronize(s));

  // After the kernel execution, copy the messages back to the host
  LogMessage_lyd* h_messages = new LogMessage_lyd;
  hipMemcpy(h_messages, d_messages, sizeof(LogMessage_lyd), hipMemcpyDeviceToHost);

  // Process and print the messages on the host
  #if PROFILE_LYD_REDUCE_BROADCAST == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | allreduce.h | runTreeUpDown | recvReduceCopy | time: %f us\n", h_messages->timeValue[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | allreduce.h | runTreeUpDown | directSendFromOutput | time: %f us\n", h_messages->timeValue1[i][0]);
  }
  #endif

  #if PROFILE_LYD_REDUCE_BROADCAST_CHUNK == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | allreduce.h | runTreeUpDown | recvReduceCopy-chunk | iteration %d | time: %f us\n", j, h_messages->timeValue[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | allreduce.h | runTreeUpDown | directSendFromOutput-chunk | iteration %d | time: %f us\n", j, h_messages->timeValue1[i][j]);
    }
  }
  #endif

  #if PROFILE_LYD_REDUCE_LOADCONN_SETDATA == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | loadRecvConn | time: %f us\n", h_messages->timeValue[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | loadSendConn | time: %f us\n", h_messages->timeValue1[i][0]);
  }

  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | primitives | setDataPtrs | time: %f us\n", h_messages->timeValue2[i][0]);
  }
  #endif

  #if PROFILE_LYD_GENERIC == 1
  for (size_t i = 0; i < maxMessages; ++i) {
    printf("DEVICE | prims_simple.h | genericop | time: %f us\n", h_messages->timeValue[i][0]);
  }
  #endif

  #if PROFILE_LYD_WAIT_REDUCE_COPY_POST
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | waitpeer | iteration %d | time: %f us\n", j, h_messages->timeValue[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | ReduceOrCopyMulti | iteration %d | time: %f us\n", j, h_messages->timeValue1[i][j]);
    }
  }
  for (size_t i = 0; i < maxMessages; ++i) {
    for (size_t j = 0; j < MAXLOGLYD; j++){
      printf("DEVICE | prims_simple.h | genericOp | postPeer | iteration %d | time: %f us\n", j, h_messages->timeValue2[i][j]);
    }
  }
  #endif

  // Free the device memory of the gauge test
  hipFree(d_messages);
  delete[] h_messages;


  //free device buffers
  CUDACHECK(hipFree(sendbuff));
  CUDACHECK(hipFree(recvbuff));


  //finalizing NCCL
  ncclCommDestroy(comm);


  //finalizing MPI
  MPICHECK(MPI_Finalize());


  printf("[MPI Rank %d] Success \n", myRank);
  return 0;
}